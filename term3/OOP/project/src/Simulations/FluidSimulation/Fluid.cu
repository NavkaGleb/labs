#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cstdlib>
#include <stdexcept>

#include "Random/Random.hpp"
#include "Fluid.cuh"

namespace ng {

#define CUDA_CALL(x) hipError_t error = hipGetLastError();                \
                     if (error != hipSuccess) {                            \
                         std::cout << hipGetErrorName(error) << std::endl; \
                         std::abort();                                      \
                     } x

    // constructor / destructor
    __host__ Fluid::Fluid(unsigned int width, unsigned int height, unsigned int scale)
            : m_size(width, height),
              m_scale(scale),
              m_elapsedTime(0.0f),
              m_timeSincePress(0.0f) {

        // init colors
        m_colors.resize(7);
        m_colors[0] = { 1.0f, 0.0f, 0.0f };
        m_colors[1] = { 0.0f, 1.0f, 0.0f };
        m_colors[2] = { 1.0f, 0.0f, 1.0f };
        m_colors[3] = { 1.0f, 1.0f, 0.0f };
        m_colors[4] = { 0.0f, 1.0f, 1.0f };
        m_colors[5] = { 1.0f, 0.0f, 1.0f };
        m_colors[6] = { 1.0f, 0.5f, 0.3f };
        m_color = m_colors[ng::Random::Irand<std::size_t>(0, m_colors.size() - 1)];

        // device memory
        const unsigned int fieldArea = m_size.x * m_size.y;
        CUDA_CALL(hipSetDevice(0));
        hipMalloc(&m_newField,    fieldArea * sizeof(kernel::Particle));
        hipMalloc(&m_oldField,    fieldArea * sizeof(kernel::Particle));
        hipMalloc(&m_newPressure, fieldArea * sizeof(float));
        hipMalloc(&m_oldPressure, fieldArea * sizeof(float));
        hipMalloc(&m_colorField,  fieldArea * 4 * sizeof(uint8_t));
        hipMalloc(&m_vorticity,   fieldArea * sizeof(float));

        // host memory
        m_pixelBuffer.resize(fieldArea * 4);

        // init SFML stuff
        m_texture.create(m_size.x, m_size.y);
    }

    __host__ Fluid::~Fluid() {
        hipFree(m_newField);
        hipFree(m_oldField);
        hipFree(m_newPressure);
        hipFree(m_oldPressure);
        hipFree(m_colorField);
        hipFree(m_vorticity);
    }

    // public methods
    __host__ void Fluid::update(float dt, const sf::Vector2i& pos1, const sf::Vector2i& pos2, bool isActive) {
        // main function, calls vorticity -> diffusion -> force -> pressure -> project -> advect -> bloom -> paint
        m_threadCount = { m_systemConfig.xThreads, m_systemConfig.yThreads };
        m_blockCount = { m_size.x / m_threadCount.x, m_size.y / m_threadCount.y };

        updateVorticity(dt);
        updateDiffusion(dt);
        updateForce(dt, pos1, pos2, isActive);
        updatePressure(dt);
        updateProjection();
        updateAdvection(dt);
        updateBloom(pos2);
        updateTexture();

        Fluid::errorCheck();
        setTexture();
    }

    // member methods
    __host__ void Fluid::errorCheck() {
        hipError_t error = hipGetLastError();

        if (error != hipSuccess)
            throw std::runtime_error(hipGetErrorName(error));
    }

    __host__ void Fluid::updateVorticity(float dt) {
        kernel::computeVorticity<<<m_blockCount, m_threadCount>>>(m_vorticity, m_oldField, m_size.x, m_size.y);
        kernel::applyVorticity<<<m_blockCount, m_threadCount>>>(
            m_newField,         m_oldField,
            m_size.x,           m_size.y,
            m_vorticity,
            m_config.vorticity, dt);
        std::swap(m_oldField, m_newField);
    }

    __host__ void Fluid::updateDiffusion(float dt) {
        // performs several iterations over velocity and color fields
        // diffuse velocity and color

        for (int i = 0; i < m_systemConfig.velocityIterations; ++i) {
            kernel::diffuse<<<m_blockCount, m_threadCount>>>(
                m_newField,                 m_oldField,
                m_size.x,                   m_size.y,
                m_config.velocityDiffusion, dt
            );
            kernel::computeColor<<<m_blockCount, m_threadCount>>>(
                m_newField,              m_oldField,
                m_size.x,                m_size.y,
                m_config.colorDiffusion, dt
            );
            std::swap(m_newField, m_oldField);
        }
    }

    __host__ void Fluid::updateForce(float dt, const sf::Vector2i& pos1, const sf::Vector2i& pos2, bool isActive) {
        // apply force
        if (!isActive) {
            m_timeSincePress += dt;
        } else {
            m_timeSincePress = 0.0f;
            m_elapsedTime += dt;

            // apply gradient to color
            auto minIndex = static_cast<unsigned int>(m_elapsedTime)     % m_colors.size();
            auto maxIndex = static_cast<unsigned int>(m_elapsedTime + 1) % m_colors.size();
            float w = m_elapsedTime - std::floor(m_elapsedTime); // int(m_elapsedTime)
            m_color = m_colors[minIndex] * (1 - w) + m_colors[maxIndex] * w;

            kernel::Vector2f force {
                static_cast<float>(pos2.x - pos1.x) * m_config.forceScale,
                static_cast<float>(pos2.y - pos1.y) * m_config.forceScale
            };
            kernel::Vector2f pos = {
                static_cast<float>(pos2.x),
                static_cast<float>(pos2.y)
            };

            kernel::applyForce<<<m_blockCount, m_threadCount>>>(
                m_oldField,
                m_size.x,        m_size.y,
                m_color,
                force,           pos,
                m_config.radius, dt
            );
        }
    }

    __host__ void Fluid::updatePressure(float dt) {
        // performs several iterations over pressure field

        for (int i = 0; i < m_systemConfig.pressureIterations; ++i) {
            kernel::computePressureImpl<<<m_blockCount, m_threadCount>>>(
                m_oldField,
                m_size.x,          m_size.y,
                m_newPressure,     m_oldPressure,
                m_config.pressure
            );

            std::swap(m_oldPressure, m_newPressure);
        }
    }

    __host__ void Fluid::updateProjection() {
        kernel::project<<<m_blockCount, m_threadCount>>>(m_oldField, m_size.x, m_size.y, m_oldPressure);
        hipMemset(m_oldPressure, 0, m_size.x * m_size.y * sizeof(float));
    }

    __host__ void Fluid::updateAdvection(float dt) {
        kernel::advect<<<m_blockCount, m_threadCount>>>(
            m_newField,                m_oldField,
            m_size.x,                  m_size.y,
            m_config.densityDiffusion, dt
        );
        std::swap(m_newField, m_oldField);
    }

    __host__ void Fluid::updateBloom(const sf::Vector2i& pos) {
        // apply bloom in mouse pos
        if (m_config.bloomEnabled && m_timeSincePress < 5.0f)
            kernel::applyBloom<<<m_blockCount, m_threadCount>>>(
                m_colorField,
                m_size.x,
                m_size.y,
                pos.x,
                pos.y,
                m_config.radius,
                m_config.bloomIntense
            );
    }

    __host__ void Fluid::updateTexture() {
        // paint image
        kernel::paint<<<m_blockCount, m_threadCount>>>(m_colorField, m_oldField, m_size.x, m_size.y);

        // copy image to cpu
        hipMemcpy(
                m_pixelBuffer.data(),
                m_colorField,
                m_size.x * m_size.y * 4 * sizeof(uint8_t),
                hipMemcpyDeviceToHost
        );
    }

    __host__ void Fluid::setTexture() {
        m_texture.update(m_pixelBuffer.data());
        m_sprite.setTexture(m_texture);
        m_sprite.setScale({ static_cast<float>(m_scale), static_cast<float>(m_scale) });
    }

    __host__ void Fluid::draw(sf::RenderTarget& target, sf::RenderStates states) const {
        target.draw(m_sprite);
    }

} // namespace ng