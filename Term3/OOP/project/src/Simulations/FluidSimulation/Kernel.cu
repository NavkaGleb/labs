#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#define SET_TO(p, x, y) if (x < width && x >= 0 && y < height && y >= 0)\
                            p = field[int(x) + int(y) * width]

#define GET_AVG(value, minValue, maxValue) min(maxValue, max(minValue, value))
#define SET_AVG(p, x, y) p = field[int(GET_AVG(x, 0.0f, width - 1.0f)) + int(GET_AVG(y, 0.0f, height - 1.0f)) * width]

namespace ng { namespace kernel {

    ////////////////////////////////////////////////////////////////////////////////////////////
    /// Vector2f
    ////////////////////////////////////////////////////////////////////////////////////////////
    // operators
    __device__ Vector2f Vector2f::operator-(const Vector2f& other) const {
        Vector2f res;

        res.x = x - other.x;
        res.y = y - other.y;

        return res;
    }

    __device__ Vector2f Vector2f::operator+(const Vector2f& other) const {
        Vector2f res;

        res.x = x + other.x;
        res.y = y + other.y;

        return res;
    }

    __device__ Vector2f Vector2f::operator*(float d) const {
        Vector2f res;

        res.x = x * d;
        res.y = y * d;

        return res;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////
    /// Color3f
    ////////////////////////////////////////////////////////////////////////////////////////////
    // operators
    __host__ __device__ Color3f Color3f::operator+(const Color3f& other) const {
        Color3f res;

        res.r = r + other.r;
        res.g = g + other.g;
        res.b = b + other.b;

        return res;
    }

    __host__ __device__ Color3f Color3f::operator*(float d) const {
        Color3f res;

        res.r = r * d;
        res.g = g * d;
        res.b = b * d;

        return res;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////
    /// Kernel Functions
    ////////////////////////////////////////////////////////////////////////////////////////////
    __device__ float curl(const Particle* field, std::size_t width, std::size_t height, int x, int y) {
        // computes curl of velocity field
        Vector2f defaultValue = field[y * width + x].velocity;
        Vector2f vec1 = { -defaultValue.x, -defaultValue.y };
        Vector2f vec2 = { -defaultValue.x, -defaultValue.y };

        SET_TO(vec1.x, x + 1, y    ).velocity.x;
        SET_TO(vec2.x, x - 1, y    ).velocity.x;
        SET_TO(vec1.y, x,     y + 1).velocity.y;
        SET_TO(vec2.y, x,     y - 1).velocity.y;

        return ((vec1.y - vec2.y) - (vec1.x - vec2.x)) * 0.5f;
    }

    __device__ Vector2f absGradient(const float* field, std::size_t width, std::size_t height, int x, int y) {
        // computes absolute value gradient of vorticity field
        float defaultValue = field[y * width + x];
        Vector2f vec1 = { defaultValue, defaultValue };
        Vector2f vec2 = { defaultValue, defaultValue };

        SET_TO(vec1.x, x + 1, y    );
        SET_TO(vec2.x, x - 1, y    );
        SET_TO(vec1.y, x,     y + 1);
        SET_TO(vec2.y, x,     y - 1);

        return { (abs(vec1.x) - abs(vec2.x)) * 0.5f, (abs(vec1.y) - abs(vec2.y)) * 0.5f };
    }

    __device__ Vector2f gradient(const float* field, std::size_t width, std::size_t height, int x, int y) {
        // computes gradient of pressure field
        float defaultValue = field[y * width + x];
        Vector2f vec1 = { defaultValue, defaultValue };
        Vector2f vec2 = { defaultValue, defaultValue };

        SET_TO(vec1.x, x + 1, y    );
        SET_TO(vec2.x, x - 1, y    );
        SET_TO(vec1.y, x,     y + 1);
        SET_TO(vec2.y, x,     y - 1);

        return { (vec1.x - vec2.x) * 0.5f, (vec1.y - vec2.y) * 0.5f };
    }

    __device__ Particle interpolate(Vector2f velocity, const Particle* field, std::size_t width, std::size_t height) {
        // interpolates quantity of grid cells
        Vector2f vec1 = { floor(velocity.x),        floor(velocity.y)        };
        Vector2f vec2 = { floor(velocity.x + 1.0f), floor(velocity.y + 1.0f) };
        Particle particles[4];

        SET_AVG(particles[0], vec1.x, vec1.y);
        SET_AVG(particles[1], vec1.x, vec2.y);
        SET_AVG(particles[2], vec2.x, vec1.y);
        SET_AVG(particles[3], vec2.x, vec2.y);

        float t1 = (vec2.x - velocity.x) / (vec2.x - vec1.x);
        float t2 = (velocity.x - vec1.x) / (vec2.x - vec1.x);
        float t3 = (vec2.y - velocity.y) / (vec2.y - vec1.y);
        float t4 = (velocity.y - vec1.y) / (vec2.y - vec1.y);

        Vector2f f1 = particles[0].velocity * t1 + particles[2].velocity * t2;
        Vector2f f2 = particles[1].velocity * t1 + particles[3].velocity * t2;

        Color3f color1 = particles[1].color * t1 + particles[3].color * t2;
        Color3f color2 = particles[1].color * t1 + particles[3].color * t2;

        return { f1 * t3 + f2 * t4, color1 * t3 + color2 * t4 };
    }

    __device__ Vector2f jacobiVelocity(const Particle* field,
                                       std::size_t width, std::size_t height,
                                       Vector2f position, Vector2f velocity,
                                       float alpha, float beta) {
        // performs iteration of jacobi method on velocity grid field
        Vector2f velocityLeft;
        Vector2f velocityRight;
        Vector2f velocityUp;
        Vector2f velocityDown;

        SET_TO(velocityLeft,  position.x - 1, position.y    ).velocity;
        SET_TO(velocityRight, position.x + 1, position.y    ).velocity;
        SET_TO(velocityUp,    position.x,     position.y - 1).velocity;
        SET_TO(velocityDown,  position.x,     position.y + 1).velocity;

        return (velocityUp + velocityDown + velocityLeft + velocityRight + velocity * alpha) * (1.0f / beta);
    }

    __device__ Color3f jacobiColor(const Particle* field,
                                   std::size_t width, std::size_t height,
                                   Vector2f pos, Color3f b,
                                   float alpha, float beta) {
        // performs iteration of jacobi method on color grid field
        Color3f colorLeft;
        Color3f colorRight;
        Color3f colorUp;
        Color3f colorDown;
        auto x = static_cast<int>(pos.x);
        auto y = static_cast<int>(pos.y);

        SET_TO(colorLeft,  x - 1, y    ).color;
        SET_TO(colorRight, x + 1, y    ).color;
        SET_TO(colorUp,    x,     y - 1).color;
        SET_TO(colorDown,  x,     y + 1).color;

        return (colorUp + colorDown + colorLeft + colorRight + b * alpha) * (1.0f / beta);
    }

    __device__ float jacobiPressure(const float* field,
                                    std::size_t width, std::size_t height,
                                    int x, int y,
                                    float b, float alpha, float beta) {
        // performs iteration of jacobi method on pressure grid field
        float defaultPressure = field[y * width + x];
        float pressureLeft = defaultPressure;
        float pressureRight = defaultPressure;
        float pressureUp = defaultPressure;
        float pressureDown = defaultPressure;

        SET_TO(pressureLeft,  x - 1, y    );
        SET_TO(pressureRight, x + 1, y    );
        SET_TO(pressureUp,    x,     y - 1);
        SET_TO(pressureDown,  x,     y + 1);

        return (pressureUp + pressureDown + pressureLeft + pressureRight + alpha * b) * (1.0f / beta);
    }

    __device__ float divergence(const Particle* field, std::size_t width, std::size_t height, int x, int y) {
        // computes divergence of velocity field
        const Particle& particle = field[y * width + x];

        Vector2f vec1 = { -particle.velocity.x, -particle.velocity.y };
        Vector2f vec2 = { -particle.velocity.x, -particle.velocity.y };

        SET_TO(vec1.x, x + 1, y    ).velocity.x;
        SET_TO(vec2.x, x - 1, y    ).velocity.x;
        SET_TO(vec1.y, x,     y + 1).velocity.y;
        SET_TO(vec2.y, x,     y - 1).velocity.y;

        return (vec1.x - vec2.x + vec1.y - vec2.y) * 0.5f;
    }

    __global__ void computeVorticity(float* vorticityField, Particle* field,
                                     std::size_t width, std::size_t height) {
        // computes vorticity field which should be passed to applyVorticity function
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        vorticityField[y * width + x] = curl(field, width, height, x, y);
    }

    __global__ void applyVorticity(Particle* newField, Particle* oldField,
                                   std::size_t width, std::size_t height,
                                   float* vorticityField, float vorticity, float dt) {
        // applies vorticity to velocity field
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        Particle& newParticle = newField[y * width + x];
        Particle& oldParticle = oldField[y * width + x];

        Vector2f newVorticity = absGradient(vorticityField, width, height, x, y);
        newVorticity.y *= -1.0f;

        float length = sqrtf(newVorticity.x * newVorticity.x + newVorticity.y * newVorticity.y) + 1e-5f;
        Vector2f vorticityNorm = newVorticity * (1.0f / length);
        Vector2f vorticityF = vorticityNorm * vorticityField[y * width + x] * vorticity;

        newParticle = oldParticle;
        newParticle.velocity = newParticle.velocity + vorticityF * dt;
    }

    __global__ void applyForce(Particle* field,
                               std::size_t width, std::size_t height,
                               Color3f color, Vector2f force, Vector2f pos, int radius, float dt) {
        // applies force and add color dye to the particle field
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        if (x >= width || y >= height)
            return;

        float e = expf(-1.0f * ((x - pos.x) * (x - pos.x) + (y - pos.y) * (y - pos.y)) / static_cast<float>(radius));
        Vector2f forceVelocity = force * dt * e;
        Particle& particle = field[y * width + x];

        particle.velocity = particle.velocity + forceVelocity;
        color = color * e + particle.color;

        particle.color.r = color.r;
        particle.color.g = color.g;
        particle.color.b = color.b;
    }

    __global__ void project(Particle* newField, std::size_t width, std::size_t height, const float* pressureField) {
        // projects pressure field on velocity field
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        Vector2f& u = newField[y * width + x].velocity;
        u = u - gradient(pressureField, width, height, x, y);
    }

    __global__ void advect(Particle* newField, Particle* oldField,
                           std::size_t width, std::size_t height,
                           float densityDiffusion, float dt) {
        // adds quantity to particles using bilinear interpolation
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        float decay = 1.0f / (1.0f + densityDiffusion * dt);
        Particle& oldParticle = oldField[y * width + x];
        Vector2f position = { static_cast<float>(x), static_cast<float>(y) };

        // find new particle tracing where it came from
        Particle newParticle = interpolate(position - oldParticle.velocity * dt, oldField, width, height);
        newParticle.velocity = newParticle.velocity * decay;

        newParticle.color.r = min(1.0f, pow(newParticle.color.r, 1.005f) * decay);
        newParticle.color.g = min(1.0f, pow(newParticle.color.g, 1.005f) * decay);
        newParticle.color.b = min(1.0f, pow(newParticle.color.b, 1.005f) * decay);

        newField[y * width + x] = newParticle;
    }

    __global__ void diffuse(Particle* newField, Particle* oldField,
                            std::size_t width, std::size_t height,
                            float velocityDiffusion, float dt) {
        // calculates nonzero divergence velocity field u
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        Vector2f velocity = oldField[y * width + x].velocity;
        Vector2f position = { static_cast<float>(x), static_cast<float>(y) };

        // perform one iteration of jacobi method (diffuse method should be called 20-50 times per cell)
        float alpha = velocityDiffusion * velocityDiffusion / dt;
        float beta = 4.0f + alpha;
        newField[y * width + x].velocity = jacobiVelocity(oldField, width, height, position, velocity, alpha, beta);
    }

    __global__ void computeColor(Particle* newField, Particle* oldField,
                                 std::size_t width, std::size_t height,
                                 float colorDiffusion, float dt) {
        // calculates color field diffusion
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        Color3f color = oldField[y * width + x].color;
        Vector2f position = { static_cast<float>(x) , static_cast<float>(y) };
        float alpha = colorDiffusion * colorDiffusion / dt;
        float beta = 4.0f + alpha;

        // perform one iteration of jacobi method (diffuse method should be called 20-50 times per cell)
        newField[y * width + x].color = jacobiColor(oldField, width, height, position, color, alpha, beta);
    }

    __global__ void computePressureImpl(Particle* field,
                                        std::size_t width, std::size_t height,
                                        float* newPressure, float* oldPressure,
                                        float pressure) {
        // performs iteration of jacobi method on pressure field
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        float div = divergence(field, width, height, x, y);
        float alpha = -1.0f * pressure * pressure;
        float beta = 4.0;

        newPressure[y * width + x] = jacobiPressure(oldPressure, width, height, x, y, div, alpha, beta);
    }

    __global__ void paint(uint8_t* colorField, const Particle* field, std::size_t width, std::size_t height) {
        // fills output image with corresponding color
        auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        if (x < width && y < height) {
            float r = field[y * width + x].color.r;
            float g = field[y * width + x].color.g;
            float b = field[y * width + x].color.b;

            colorField[4 * (y * width + x) + 0] = min(255.0f, 255.0f * r);
            colorField[4 * (y * width + x) + 1] = min(255.0f, 255.0f * g);
            colorField[4 * (y * width + x) + 2] = min(255.0f, 255.0f * b);
            colorField[4 * (y * width + x) + 3] = 255;
        }
    }

    __global__ void applyBloom(uint8_t* colorField,
                               std::size_t width, std::size_t height,
                               int positionX, int positionY,
                               float radius, float bloomIntense) {
        // adds flashlight effect near the mouse position
        int x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
        int y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

        if (x >= width || y >= height)
            return;

        int pos = static_cast<int>(y * width + x) * 4;
        float factor = bloomIntense * expf(-(
                static_cast<float>((x - positionX) * (x - positionX) +
                                   (y - positionY) * (y - positionY)) + 1.0f
        ) / (radius * radius));

        float r = colorField[pos + 0];
        float g = colorField[pos + 1];
        float b = colorField[pos + 2];

        float maxValue = max(r, max(g, b));

        colorField[pos + 0] = min(255.0f, r + maxValue * factor);
        colorField[pos + 1] = min(255.0f, g + maxValue * factor);
        colorField[pos + 2] = min(255.0f, b + maxValue * factor);
    }

} } // namespace ng::kernel